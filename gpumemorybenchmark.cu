#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  

#include <stdio.h>
#include <conio.h>


__global__ void memoryBenc(int *g_out, int *g_in, int N, int inner_reps)
{
	
    int idx = blockIdx.x * blockDim.x ;
	int threadx = idx + threadIdx.x;
    if (threadx > N){
		int a = 10;
	}
	else {
        for (int i=0; i<inner_reps; ++i)
        {
            g_out[threadx] = g_in[threadx] + 1;
        }
    }
}

#define STREAM_COUNT 4

int *h_data_source;
int *h_data_sink;

int *h_data_in[STREAM_COUNT];
int *d_data_in[STREAM_COUNT];

int *h_data_out[STREAM_COUNT];
int *d_data_out[STREAM_COUNT];


hipEvent_t cycleDone[STREAM_COUNT];
hipStream_t stream[STREAM_COUNT];

hipEvent_t start, stop;

int N = 1 << 22;

//taking an average of 25 operations

int nreps = 25;           
int inner_reps = 5;

int memsize;

dim3 block(512);
dim3 grid;

int thread_blocks;

float processWithStreams(int streams_used);

std::string getRandOut() {
	std::string randOut = "sequential multi stream read write with ^ streams in GBps =";
	return randOut;
}

std::string getdOut() {
	std::string randOut = "sequential read write with single stream in GBps =";
	return randOut;
}

int setDev() {
	return 0;
}


int main(int argc, char *argv[])
{
	hipError_t error;
	float test_blocks = 1500;
	int cuda_device = setDev();
	cuda_device = 0;
	float scale_factor;
    hipDeviceProp_t deviceProp;
	const char *name = "simpleMultiCopy";

		error = hipGetDeviceProperties(&deviceProp, 0);

	std::cout<<"size   of array = "<<N<<"\n";

    memsize = N * sizeof(int);

    thread_blocks = N / block.x;

    grid.x = thread_blocks % 65535;
	int val = 65535 + 1;
    grid.y = (thread_blocks / val);


    h_data_source = (int *) malloc(memsize);
    h_data_sink = (int *) malloc(memsize);

    for (int i =0; i<STREAM_COUNT; ++i)
    {

		error = hipHostAlloc(&h_data_in[i], memsize,
                                      hipHostMallocDefault);
		error = hipMalloc(&d_data_in[i], memsize);

		error = hipHostAlloc(&h_data_out[i], memsize,
                                      hipHostMallocDefault);
		error = hipMalloc(&d_data_out[i], memsize);

		error = hipStreamCreate(&stream[i]);
		error = hipEventCreate(&cycleDone[i]);

        hipEventRecord(cycleDone[i], stream[i]);
    }

    hipEventCreate(&start);

	float memcpy_h2d_time; 
	hipEventCreate(&stop);

	for (int i = 0; i<N; ++i)
	{
		h_data_source[i] = 0;
	}

	for (int i = 0; i<STREAM_COUNT; ++i)
	{
		memcpy(h_data_in[i], h_data_source, memsize);
	}
	hipEventRecord(start,0);
	error = hipMemcpyAsync(d_data_in[0], h_data_in[0], memsize,
                                    hipMemcpyHostToDevice,0);
    hipEventRecord(stop,0);

	double kt=0.00;
	hipEventSynchronize(stop);

	long kernelTime = 0;

    hipEventElapsedTime(&memcpy_h2d_time, start, stop);

    hipEventRecord(start,0);
	error = hipMemcpyAsync(h_data_out[0], d_data_out[0], memsize,
                                    hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float memcpy_d2h_time;
    hipEventElapsedTime(&memcpy_d2h_time, start, stop);

    hipEventRecord(start,0);
    memoryBenc<<<grid, block,0,0>>>(d_data_out[0], d_data_in[0], N, inner_reps);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

	float time1 = (memsize * 1e-6) / memcpy_h2d_time;

	std::cout << "write throughput in GBps= "<<time1<<std::endl;

	float time2 = (memsize * 1e-6) / memcpy_d2h_time;
	std::cout << "read throughput in GBps= " << time2 << std::endl;

    float serial_time = processWithStreams(1);
    float overlap_time = processWithStreams(STREAM_COUNT);
	 
	std::cout << "\nread write outputs - \n" << std::endl;

	std::string serialOut = getdOut();
	float temp1 = (memsize * 2e-6) / serial_time;
	float seqOut = nreps * temp1;

	std::cout << serialOut << seqOut<<std::endl;
	
    free(h_data_source);
	std::cout << "streams used = " << STREAM_COUNT << "\n";
	std::string randOut = getRandOut();
	free(h_data_sink);
	float temp2 = (memsize * 2e-6) / overlap_time;
	float randOutval = nreps * temp2;
	hipHostFree(h_data_in[1]);
		hipFree(d_data_in[1]);

		hipHostFree(h_data_out[1]);
		hipFree(d_data_out[1]);

		hipStreamDestroy(stream[1]);
		hipEventDestroy(cycleDone[1]);

		hipHostFree(h_data_in[2]);
		hipFree(d_data_in[2]);

		hipHostFree(h_data_out[2]);
		hipFree(d_data_out[2]);

		hipStreamDestroy(stream[2]);
		hipEventDestroy(cycleDone[2]);

		hipHostFree(h_data_in[3]);
		hipFree(d_data_in[3]);

		hipHostFree(h_data_out[3]);
		hipFree(d_data_out[3]);

		hipStreamDestroy(stream[3]);
		hipEventDestroy(cycleDone[3]);

		hipHostFree(h_data_in[0]);
		hipFree(d_data_in[0]);

		hipHostFree(h_data_out[0]);
		hipFree(d_data_out[0]);

		hipStreamDestroy(stream[0]);
		hipEventDestroy(cycleDone[0]);


	std::cout << randOut << randOutval << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

	getch();

    exit(EXIT_SUCCESS);
}

float processWithStreams(int streams_used)
{

    int current_stream = 0;

    float time;

	std::string tpy = "this is similar to threading in cpu";

    hipEventRecord(start, 0);

    for (int i=0; i<nreps; ++i)
    {
        int next_stream = (current_stream + 1) % streams_used;


		hipError_t error =hipEventSynchronize(cycleDone[next_stream]);

        memoryBenc<<<grid, block, 0, stream[current_stream]>>>(
            d_data_out[current_stream],
            d_data_in[current_stream],
            N,
            inner_reps);
        checkCudaErrors(hipMemcpyAsync(
                            d_data_in[next_stream],
                            h_data_in[next_stream],
                            memsize,
                            hipMemcpyHostToDevice,
                            stream[next_stream]));

        checkCudaErrors(hipMemcpyAsync(
                            h_data_out[current_stream],
                            d_data_out[current_stream],
                            memsize,
                            hipMemcpyDeviceToHost,
                            stream[current_stream]));

        checkCudaErrors(hipEventRecord(
                            cycleDone[current_stream],
                            stream[current_stream]));
        current_stream = next_stream;
    }
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    return time;
}
//reference: simplemulticopy form the nvidia samples (installation examples)